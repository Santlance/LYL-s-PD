#include "hip/hip_runtime.h"
#include "pd_cuda.h"

#define GRAVITY -10.0
//#define GRAVITY 0

int vertex_number_cuda;				// ������Ŀ
int tet_number_cuda;				// ��������Ŀ

float* positions_cuda;				// λ��
float* last_positions_cuda;			// ��һ��λ�ã���һ֡��λ�ã�
float* old_positions_cuda;			// sn
float* prev_positions_cuda;			// ��һ�ε�����λ��
float* next_positions_cuda;			// ��һ��λ��
float* velocity_cuda;				// �ٶ�
float* external_force_cuda;			// ����
float* mass_cuda;					// ����

float* volumeDiag_cuda;				// �Խ�Ԫ��
float* tetInvD3x3_cuda;				// 
float* tetInvD3x4_cuda;
float* tetVolume_cuda;
int* tet_indices_cuda;				// ����������
unsigned int* tet_draw_indices_cuda;
float* tet_stiffness_cuda;			// �����嵯��ϵ��

float* force_cuda;					// ��������
float* fixed_cuda;					// �����Ƿ�̶�

// λ��Լ��
int pos_constraint_num_cuda;				// Լ������
unsigned int* pos_constraint_indices_cuda;	// ����
float* pos_constraint_targets_cuda;			// Ŀ��λ��

// surface
unsigned int face_number_cuda;
unsigned int* tet_faces_indices_cuda;		// ��������������
float* normals_cuda;						// ÿ������ķ���


// OpenGL
hipGraphicsResource* tet_positions_gl;
hipGraphicsResource* tet_indices_gl;
hipGraphicsResource* tet_face_indices_gl;
hipGraphicsResource* tet_normals_gl;


/*
* ע�� EigenMatrixX.data() ������Ķ�Ӧ��ϵ��һ��һ�еģ�Ҳ����˵
*	EigenMatrixX.data() = [1, 2, 3, 4, 5, 6, 7, 8, 9]
* ��Ӧ�ľ���Ϊ
*		[1, 4, 7]
*		[2, 5, 8]
*		[3, 6, 9]
*/


// ------------------------------------------------------------------------------
// ------------------------- CUDA �˺��� ----------------------------------------
// ------------------------------------------------------------------------------

// *************************** device ******************************

// R = A * B
__device__ void matrix_product_33(float* R, float* A, float* B)
{
	R[0] = A[0] * B[0] + A[1] * B[3] + A[2] * B[6];
	R[1] = A[0] * B[1] + A[1] * B[4] + A[2] * B[7];
	R[2] = A[0] * B[2] + A[1] * B[5] + A[2] * B[8];
	R[3] = A[3] * B[0] + A[4] * B[3] + A[5] * B[6];
	R[4] = A[3] * B[1] + A[4] * B[4] + A[5] * B[7];
	R[5] = A[3] * B[2] + A[4] * B[5] + A[5] * B[8];
	R[6] = A[6] * B[0] + A[7] * B[3] + A[8] * B[6];
	R[7] = A[6] * B[1] + A[7] * B[4] + A[8] * B[7];
	R[8] = A[6] * B[2] + A[7] * B[5] + A[8] * B[8];
}

// R = A * B
__device__ void matrix_product(float* R, float* A, float* B, int m, int n, int p)
{
	memset(R, 0, sizeof(float) * m * p);
	for (int i = 0; i < m; i++)
		for (int j = 0; j < p; j++)
			for (int k = 0; k < n; k++)
				R[i * p + j] += A[i * n + k] * B[k * p + j];
}

// R = A - B
__device__ void matrix_substract_33(float* R, float* A, float* B)
{
	R[0] = A[0] - B[0];
	R[1] = A[1] - B[1];
	R[2] = A[2] - B[2];
	R[3] = A[3] - B[3];
	R[4] = A[4] - B[4];
	R[5] = A[5] - B[5];
	R[6] = A[6] - B[6];
	R[7] = A[7] - B[7];
	R[8] = A[8] - B[8];
}

// ���α��ݶ�F��ȡR
__device__ void get_R_from_F(float F[3][3], float R[3][3])
{
	float C[3][3];
	memset(&C[0][0], 0, sizeof(float) * 9);
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			for (int k = 0; k < 3; k++)
				C[i][j] += F[k][i] * F[k][j];

	float C2[3][3];
	memset(&C2[0][0], 0, sizeof(float) * 9);
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			for (int k = 0; k < 3; k++)
				C2[i][j] += C[i][k] * C[j][k];

	float det = F[0][0] * F[1][1] * F[2][2] +
		F[0][1] * F[1][2] * F[2][0] +
		F[1][0] * F[2][1] * F[0][2] -
		F[0][2] * F[1][1] * F[2][0] -
		F[0][1] * F[1][0] * F[2][2] -
		F[0][0] * F[1][2] * F[2][1];

	float I_c = C[0][0] + C[1][1] + C[2][2];
	float I_c2 = I_c * I_c;
	float II_c = 0.5 * (I_c2 - C2[0][0] - C2[1][1] - C2[2][2]);
	float III_c = det * det;
	float k = I_c2 - 3 * II_c;

	float inv_U[3][3];
	if (k < 1e-10f)
	{
		float inv_lambda = 1 / sqrt(I_c / 3);
		memset(inv_U, 0, sizeof(float) * 9);
		inv_U[0][0] = inv_lambda;
		inv_U[1][1] = inv_lambda;
		inv_U[2][2] = inv_lambda;
	}
	else
	{
		float l = I_c * (I_c * I_c - 4.5 * II_c) + 13.5 * III_c;
		float k_root = sqrt(k);
		float value = l / (k * k_root);
		if (value < -1.0) value = -1.0;
		if (value > 1.0) value = 1.0;
		float phi = acos(value);
		float lambda2 = (I_c + 2 * k_root * cos(phi / 3)) / 3.0;
		float lambda = sqrt(lambda2);

		float III_u = sqrt(III_c);
		if (det < 0)   III_u = -III_u;
		float I_u = lambda + sqrt(-lambda2 + I_c + 2 * III_u / lambda);
		float II_u = (I_u * I_u - I_c) * 0.5;

		float U[3][3];
		float inv_rate, factor;

		inv_rate = 1 / (I_u * II_u - III_u);
		factor = I_u * III_u * inv_rate;

		memset(U, 0, sizeof(float) * 9);
		U[0][0] = factor;
		U[1][1] = factor;
		U[2][2] = factor;

		factor = (I_u * I_u - II_u) * inv_rate;
		for (int i = 0; i < 3; i++)
			for (int j = 0; j < 3; j++)
				U[i][j] += factor * C[i][j] - inv_rate * C2[i][j];

		inv_rate = 1 / III_u;
		factor = II_u * inv_rate;
		memset(inv_U, 0, sizeof(float) * 9);
		inv_U[0][0] = factor;
		inv_U[1][1] = factor;
		inv_U[2][2] = factor;


		factor = -I_u * inv_rate;
		for (int i = 0; i < 3; i++)
			for (int j = 0; j < 3; j++)
				inv_U[i][j] += factor * U[i][j] + inv_rate * C[i][j];
	}

	memset(&R[0][0], 0, sizeof(float) * 9);
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			for (int k = 0; k < 3; k++)
				R[i][j] += F[i][k] * inv_U[k][j];

	//��飬����invert
	if (det <= 0)
	{
		R[0][0] = 1;
		R[0][1] = 0;
		R[0][2] = 0;
		R[1][0] = 0;
		R[1][1] = 1;
		R[1][2] = 0;
		R[2][0] = 0;
		R[2][1] = 0;
		R[2][2] = 1;
	}
}

// ***********************************************************



// *************************** global ******************************

__global__ void calculate_Sn(float* positions, float* velocity, float* external_force,
	float* old_positions, float* prev_positions, float* last_positions, float* fixed,
	int vertices_number, float damping, float dt)
{
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadId >= vertices_number) return;

	int idx = threadId * 3 + 0;
	int idy = threadId * 3 + 1;
	int idz = threadId * 3 + 2;

	last_positions[idx] = positions[idx];
	last_positions[idy] = positions[idy];
	last_positions[idz] = positions[idz];

	// �̶��㲻�˶�
	if (fixed[threadId] < 10.0)
	{
		// �˶�����
		velocity[idx] *= damping;
		velocity[idy] *= damping;
		velocity[idz] *= damping;

		// �������
		velocity[idy] += GRAVITY * dt;

		// ��������
		velocity[idx] += external_force[idx] * dt;
		velocity[idy] += external_force[idy] * dt;
		velocity[idz] += external_force[idz] * dt;

		// λ��Ԥ��
		positions[idx] += velocity[idx] * dt;
		positions[idy] += velocity[idy] * dt;
		positions[idz] += velocity[idz] * dt;
	}

	// old_positions ��¼ Sn
	old_positions[idx] = positions[idx];
	old_positions[idy] = positions[idy];
	old_positions[idz] = positions[idz];
	prev_positions[idx] = positions[idx];
	prev_positions[idy] = positions[idy];
	prev_positions[idz] = positions[idz];

	// �������㣬����Ӧ��ÿʱ�̸���
	external_force[idx] = 0;
	external_force[idy] = 0;
	external_force[idz] = 0;
}

__global__ void calculate_tet(float* positions, int* tetIndex, float* tetInvD3x3,
	float* tetInvD3x4, float* force, float* tet_volume, int tet_number, float* tet_stiffness)
{
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadId >= tet_number) return;

	// TODO active[]

	int id0 = tetIndex[threadId * 4 + 0];
	int id1 = tetIndex[threadId * 4 + 1];
	int id2 = tetIndex[threadId * 4 + 2];
	int id3 = tetIndex[threadId * 4 + 3];

	// shape ����
	float D[9];

	D[0] = positions[id1 * 3 + 0] - positions[id0 * 3 + 0];
	D[1] = positions[id2 * 3 + 0] - positions[id0 * 3 + 0];
	D[2] = positions[id3 * 3 + 0] - positions[id0 * 3 + 0];
	D[3] = positions[id1 * 3 + 1] - positions[id0 * 3 + 1];
	D[4] = positions[id2 * 3 + 1] - positions[id0 * 3 + 1];
	D[5] = positions[id3 * 3 + 1] - positions[id0 * 3 + 1];
	D[6] = positions[id1 * 3 + 2] - positions[id0 * 3 + 2];
	D[7] = positions[id2 * 3 + 2] - positions[id0 * 3 + 2];
	D[8] = positions[id3 * 3 + 2] - positions[id0 * 3 + 2];

	// �����α��ݶ�F
	float F[9];
	matrix_product_33(F, D, &tetInvD3x3[threadId * 9]);

	// �� F �зֽ�� R���α��ݶȷֽ����ת�ɷ֣�
	float R[9];
	get_R_from_F((float(*)[3])F, (float(*)[3])R);

	// R = R - F
	matrix_substract_33(R, R, F);

	// force = - w * Ac^T * (F - R)	�������󵼿ɵ�
	float temp[12];
	matrix_product(temp, R, &tetInvD3x4[threadId * 12], 3, 3, 4);
	float coef = tet_volume[threadId] * tet_stiffness[threadId];
	// ԭ�Ӳ�������������������
	atomicAdd(force + id0 * 3 + 0, temp[0] * coef);
	atomicAdd(force + id0 * 3 + 1, temp[4] * coef);
	atomicAdd(force + id0 * 3 + 2, temp[8] * coef);
	atomicAdd(force + id1 * 3 + 0, temp[1] * coef);
	atomicAdd(force + id1 * 3 + 1, temp[5] * coef);
	atomicAdd(force + id1 * 3 + 2, temp[9] * coef);
	atomicAdd(force + id2 * 3 + 0, temp[2] * coef);
	atomicAdd(force + id2 * 3 + 1, temp[6] * coef);
	atomicAdd(force + id2 * 3 + 2, temp[10] * coef);
	atomicAdd(force + id3 * 3 + 0, temp[3] * coef);
	atomicAdd(force + id3 * 3 + 1, temp[7] * coef);
	atomicAdd(force + id3 * 3 + 2, temp[11] * coef);
}

__global__ void calculate_position(float* positions, float* force, float* mass,
	float* next_positions, float* prev_positions, float* old_positions, float* volumeDiag, float* fixed,
	int vertices_number, float dt, float omega)
{
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadId >= vertices_number) return;

	//

	int idx = threadId * 3 + 0;
	int idy = threadId * 3 + 1;
	int idz = threadId * 3 + 2;

	// M / (h^2)
	float diagConstant = (mass[threadId] + fixed[threadId]) / (dt * dt);

	// shape match?

	// x_n1 = D^-1 * (M / h^2 * (Sn - x_n) + Sigma(w * A^T *(R - F))) + x_n  ���շ�������
	next_positions[idx] = (diagConstant * (old_positions[idx] - positions[idx]) + force[idx]) / (volumeDiag[threadId] + diagConstant) + positions[idx];
	next_positions[idy] = (diagConstant * (old_positions[idy] - positions[idy]) + force[idy]) / (volumeDiag[threadId] + diagConstant) + positions[idy];
	next_positions[idz] = (diagConstant * (old_positions[idz] - positions[idz]) + force[idz]) / (volumeDiag[threadId] + diagConstant) + positions[idz];

	// ��������
	force[idx] = force[idy] = force[idz] = 0;

	// �б�ѩ�����
	next_positions[idx] = (next_positions[idx] - positions[idx]) * 0.6 + positions[idx];
	next_positions[idy] = (next_positions[idy] - positions[idy]) * 0.6 + positions[idy];
	next_positions[idz] = (next_positions[idz] - positions[idz]) * 0.6 + positions[idz];

	next_positions[idx] = omega * (next_positions[idx] - prev_positions[idx]) + prev_positions[idx];
	next_positions[idy] = omega * (next_positions[idy] - prev_positions[idy]) + prev_positions[idy];
	next_positions[idz] = omega * (next_positions[idz] - prev_positions[idz]) + prev_positions[idz];

	// ����
	prev_positions[idx] = positions[idx];
	prev_positions[idy] = positions[idy];
	prev_positions[idz] = positions[idz];

	positions[idx] = next_positions[idx];
	positions[idy] = next_positions[idy];
	positions[idz] = next_positions[idz];
}

__global__ void calculate_velocity(float* positions, float* velocity,
	float* last_positions, int vertices_number, float dt)
{
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadId >= vertices_number) return;

	int idx = threadId * 3 + 0;
	int idy = threadId * 3 + 1;
	int idz = threadId * 3 + 2;

	// �����ٶ�
	velocity[idx] = (positions[idx] - last_positions[idx]) / dt;
	velocity[idy] = (positions[idy] - last_positions[idy]) / dt;
	velocity[idz] = (positions[idz] - last_positions[idz]) / dt;

	// ������ʱ��ôд��������ģ��һ��ƽ��
	//float ground = -10;
	float ground = -0.402;
	if (positions[idy] < ground)
	{
		positions[idy] = ground;

		velocity[idx] = 0;
		velocity[idy] = -velocity[idy];
		velocity[idz] = 0;
	}
}

__global__ void calculate_position_constraint_force(float* position, float* force, int constraint_number,
	unsigned int* constraint_indices, float* constraint_target, int stiffness)
{
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadId >= constraint_number) return;

	// Լ�����id
	unsigned int id = constraint_indices[threadId];

	// Լ��������
	float px = position[id * 3 + 0];
	float py = position[id * 3 + 1];
	float pz = position[id * 3 + 2];
	// Ŀ�������
	float tx = constraint_target[threadId * 3 + 0];
	float ty = constraint_target[threadId * 3 + 1];
	float tz = constraint_target[threadId * 3 + 2];

	// Լ���㵽Ŀ�����˶�����
	float dx = tx - px;
	float dy = ty - py;
	float dz = tz - pz;

	// �Ծ���Ϊϵ���������
	atomicAdd(force + id * 3 + 0, dx * stiffness);
	atomicAdd(force + id * 3 + 1, dy * stiffness);
	atomicAdd(force + id * 3 + 2, dz * stiffness);
}


__global__ void clear_normal(float* normal, int vertex_number)
{
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadId >= vertex_number) return;

	normal[threadId * 3 + 0] = 0;
	normal[threadId * 3 + 1] = 0;
	normal[threadId * 3 + 2] = 0;
}

__global__ void update_normal(float* position, float* normal, unsigned int* face_indices, int face_num)
{
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadId >= face_num) return;

	unsigned int id0 = face_indices[threadId * 3 + 0];
	unsigned int id1 = face_indices[threadId * 3 + 1];
	unsigned int id2 = face_indices[threadId * 3 + 2];

	// ����������������
	float ax = position[id1 * 3 + 0] - position[id0 * 3 + 0];
	float ay = position[id1 * 3 + 1] - position[id0 * 3 + 1];
	float az = position[id1 * 3 + 2] - position[id0 * 3 + 2];
	float bx = position[id2 * 3 + 0] - position[id0 * 3 + 0];
	float by = position[id2 * 3 + 1] - position[id0 * 3 + 1];
	float bz = position[id2 * 3 + 2] - position[id0 * 3 + 2];

	// ��˼��㷨��
	float crossx = ay * bz - by * az;
	float crossy = az * bx - bz * ax;
	float crossz = ax * by - bx * ay;

	// ���ߵ�λ��
	float len = sqrt(crossx * crossx + crossy * crossy + crossz * crossz);
	crossx /= len;
	crossy /= len;
	crossz /= len;

	// �����ۼӵ�������
	atomicAdd(normal + id0 * 3 + 0, crossx);
	atomicAdd(normal + id0 * 3 + 1, crossy);
	atomicAdd(normal + id0 * 3 + 2, crossz);
	atomicAdd(normal + id1 * 3 + 0, crossx);
	atomicAdd(normal + id1 * 3 + 1, crossy);
	atomicAdd(normal + id1 * 3 + 2, crossz);
	atomicAdd(normal + id2 * 3 + 0, crossx);
	atomicAdd(normal + id2 * 3 + 1, crossy);
	atomicAdd(normal + id2 * 3 + 2, crossz);
}

__global__ void normalize_normal(float* normal, int vertex_number)
{
	unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadId >= vertex_number) return;

	float nx = normal[threadId * 3 + 0];
	float ny = normal[threadId * 3 + 1];
	float nz = normal[threadId * 3 + 2];

	float len = sqrt(nx * nx + ny * ny + nz * nz);
	normal[threadId * 3 + 0] = nx / len;
	normal[threadId * 3 + 1] = ny / len;
	normal[threadId * 3 + 2] = nz / len;
}

// ***********************************************************

// ------------------------------------------------------------------------------
// ------------------------------------------------------------------------------
// ------------------------------------------------------------------------------




void cuda_calculate_Sn(float damping, float dt)
{
	// ÿ��update�����ȵ�����������������Ҫ��CUDA��OpenGL�Ĺ�����
	hipGraphicsMapResources(1, &tet_positions_gl, 0);
	size_t size = vertex_number_cuda * 3 * sizeof(float);
	hipGraphicsResourceGetMappedPointer((void**)&positions_cuda, &size, tet_positions_gl);		// ��ȡ�ڴ�ָ��

	hipGraphicsMapResources(1, &tet_indices_gl, 0);
	size = tet_number_cuda * 12 * sizeof(unsigned int);
	hipGraphicsResourceGetMappedPointer((void**)&tet_draw_indices_cuda, &size, tet_indices_gl);

	// ����
	int block_number = (vertex_number_cuda + CUDA_THREAD_NUMBER - 1) / CUDA_THREAD_NUMBER;

	calculate_Sn << <block_number, CUDA_THREAD_NUMBER >> > (
		positions_cuda, velocity_cuda, external_force_cuda, old_positions_cuda,
		prev_positions_cuda, last_positions_cuda, fixed_cuda, vertex_number_cuda, damping, dt);

	// ͬ��
	hipDeviceSynchronize();
}

void cuda_calculate_tet()
{
	int block_number = (tet_number_cuda + CUDA_THREAD_NUMBER - 1) / CUDA_THREAD_NUMBER;

	calculate_tet << < block_number, CUDA_THREAD_NUMBER >> > (
		positions_cuda, tet_indices_cuda, tetInvD3x3_cuda, tetInvD3x4_cuda,
		force_cuda, tetVolume_cuda, tet_number_cuda, tet_stiffness_cuda);

	hipDeviceSynchronize();
}

void cuda_calculate_positions(float omega, float dt)
{
	int block_number = (vertex_number_cuda + CUDA_THREAD_NUMBER - 1) / CUDA_THREAD_NUMBER;

	calculate_position << <block_number, CUDA_THREAD_NUMBER >> > (
		positions_cuda, force_cuda, mass_cuda, next_positions_cuda, prev_positions_cuda,
		old_positions_cuda, volumeDiag_cuda, fixed_cuda, vertex_number_cuda, dt, omega
		);

	hipDeviceSynchronize();
}

void cuda_calculate_velocity(float dt)
{
	int block_number = (vertex_number_cuda + CUDA_THREAD_NUMBER - 1) / CUDA_THREAD_NUMBER;

	calculate_velocity << <block_number, CUDA_THREAD_NUMBER >> > (
		positions_cuda, velocity_cuda, last_positions_cuda, vertex_number_cuda, dt
		);

	// ����PD�����һ����Ҫ����󶨣�����OpenGL�޷���Ⱦ
	hipGraphicsUnmapResources(1, &tet_indices_gl, 0);
	hipGraphicsUnmapResources(1, &tet_positions_gl, 0);

	hipDeviceSynchronize();
}

void cuda_calculate_position_constraint_force(int stiffness)
{
	int block_number = (pos_constraint_num_cuda + CUDA_THREAD_NUMBER - 1) / CUDA_THREAD_NUMBER;

	calculate_position_constraint_force << <block_number, CUDA_THREAD_NUMBER >> > (
		positions_cuda, force_cuda, pos_constraint_num_cuda,
		pos_constraint_indices_cuda, pos_constraint_targets_cuda, stiffness
		);

	hipDeviceSynchronize();
}

void cuda_update_normals()
{
	// map
	hipGraphicsMapResources(1, &tet_positions_gl, 0);
	size_t size = vertex_number_cuda * 3 * sizeof(float);
	hipGraphicsResourceGetMappedPointer((void**)&positions_cuda, &size, tet_positions_gl);		// ��ȡ�ڴ�ָ��

	hipGraphicsMapResources(1, &tet_normals_gl, 0);
	hipGraphicsResourceGetMappedPointer((void**)&normals_cuda, &size, tet_normals_gl);

	// update
	int block_number = (vertex_number_cuda + CUDA_THREAD_NUMBER - 1) / CUDA_THREAD_NUMBER;
	clear_normal << <block_number, CUDA_THREAD_NUMBER >> > (normals_cuda, vertex_number_cuda);

	block_number= (face_number_cuda + CUDA_THREAD_NUMBER - 1) / CUDA_THREAD_NUMBER;
	update_normal << <block_number, CUDA_THREAD_NUMBER >> > (positions_cuda, normals_cuda, tet_faces_indices_cuda, face_number_cuda);
	
	block_number = (vertex_number_cuda + CUDA_THREAD_NUMBER - 1) / CUDA_THREAD_NUMBER;
	normalize_normal << <block_number, CUDA_THREAD_NUMBER >> > (normals_cuda, vertex_number_cuda);

	// unmap
	hipGraphicsUnmapResources(1, &tet_indices_gl, 0);
	hipGraphicsUnmapResources(1, &tet_normals_gl, 0);
	hipDeviceSynchronize();
}
